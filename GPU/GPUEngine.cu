#include "hip/hip_runtime.h"
/*
 * This file is part of the BTCCollider distribution (https://github.com/JeanLucPons/Kangaroo).
 * Copyright (c) 2020 Jean Luc PONS.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, version 3.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#ifndef WIN64
#include <unistd.h>
#include <stdio.h>
#endif

#include "GPUEngine.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <cassert> // For assert function
#include "../Timer.h"

#include "GPUMath.h"
#include "GPUCompute.h"
#include "../CudaErrorHandler.h" // Use the comprehensive error handler from root directory

// Use modern CUDA error handling system
// Legacy CheckCudaError function replaced by CudaErrorChecker class

// Compatibility wrapper for legacy CheckCudaError function
bool CheckCudaError(hipError_t err, const char *message, bool exitOnError = false, const char *context = nullptr, int index = -1)
{
  if (err != hipSuccess)
  {
    std::string full_message = std::string(message);
    if (context && index >= 0)
    {
      full_message += " (" + std::string(context) + " " + std::to_string(index) + ")";
    }

    printf("CUDA Error: %s - %s\n", full_message.c_str(), hipGetErrorString(err));

    if (exitOnError)
    {
      exit(1);
    }
    return false;
  }
  return true;
}

// ---------------------------------------------------------------------------------------

__global__ void comp_kangaroos(uint64_t *kangaroos, uint32_t maxFound, uint32_t *found, uint64_t dpMask)
{

  int xPtr = (blockIdx.x * blockDim.x * GPU_GRP_SIZE) * KSIZE; // x[4] , y[4] , d[2], lastJump
  ComputeKangaroos(kangaroos + xPtr, maxFound, found, dpMask);
}

// ---------------------------------------------------------------------------------------
// #define GPU_CHECK
#ifdef GPU_CHECK
__global__ void check_gpu()
{

  // Check ModInv
  uint64_t N[5] = {0x0BE3D7593BE1147CULL, 0x4952AAF512875655ULL, 0x08884CCAACCB9B53ULL, 0x9EAE2E2225044292ULL, 0ULL};
  uint64_t I[5];
  uint64_t R[5];
  bool ok = true;

  /*
  for(uint64_t i=0;i<10000 && ok;i++) {

    Load(R,N);
    _ModInv(R);
    Load(I,R);
    _ModMult(R,N);
    SubP(R);
    if(!_IsOne(R)) {
      ok = false;
      printf("ModInv wrong %d\n",(int)i);
      printf("N = %016llx %016llx %016llx %016llx %016llx\n",N[4],N[3],N[2],N[1],N[0]);
      printf("I = %016llx %016llx %016llx %016llx %016llx\n",I[4],I[3],I[2],I[1],I[0]);
      printf("R = %016llx %016llx %016llx %016llx %016llx\n",R[4],R[3],R[2],R[1],R[0]);
    }

    N[0]++;

  }
  */
  I[4] = 0;
  R[4] = 0;
  for (uint64_t i = 0; i < 100000 && ok; i++)
  {

    _ModSqr(I, N);
    _ModMult(R, N, N);
    if (!_IsEqual(I, R))
    {
      ok = false;
      printf("_ModSqr wrong %d\n", (int)i);
      printf("N = %016llx %016llx %016llx %016llx %016llx\n", N[4], N[3], N[2], N[1], N[0]);
      printf("I = %016llx %016llx %016llx %016llx %016llx\n", I[4], I[3], I[2], I[1], I[0]);
      printf("R = %016llx %016llx %016llx %016llx %016llx\n", R[4], R[3], R[2], R[1], R[0]);
    }

    N[0]++;
  }
}
#endif

// ---------------------------------------------------------------------------------------

using namespace std;

int _ConvertSMVer2Cores(int major, int minor)
{

  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct
  {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version,
            // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = {
      {0x20, 32}, // Fermi Generation (SM 2.0) GF100 class
      {0x21, 48}, // Fermi Generation (SM 2.1) GF10x class
      {0x30, 192},
      {0x32, 192},
      {0x35, 192},
      {0x37, 192},
      {0x50, 128},
      {0x52, 128},
      {0x53, 128},
      {0x60, 64},
      {0x61, 128},
      {0x62, 128},
      {0x70, 64},
      {0x72, 64},
      {0x75, 64},
      {-1, -1}};

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1)
  {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
    {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  return 0;
}

void GPUEngine::SetWildOffset(Int *offset)
{
  wildOffset.Set(offset);
}

GPUEngine::GPUEngine(int nbThreadGroup, int nbThreadPerGroup, int gpuId, uint32_t maxFound)
{

  // Initialise CUDA
  this->nbThreadPerGroup = nbThreadPerGroup;
  initialised = false;
  hipError_t err;

  int deviceCount = 0;
  hipDeviceProp_t deviceProp;

  err = hipGetDeviceCount(&deviceCount);
  if (!CheckCudaError(err, "hipGetDeviceCount"))
  {
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  err = hipSetDevice(gpuId);
  if (!CheckCudaError(err, "hipSetDevice"))
  {
    return;
  }

  err = hipGetDeviceProperties(&deviceProp, gpuId);
  if (!CheckCudaError(err, "hipGetDeviceProperties"))
  {
    return;
  }

  this->nbThread = nbThreadGroup * nbThreadPerGroup;
  this->maxFound = maxFound;
  outputSize = (maxFound * ITEM_SIZE + 4);

  char tmp[512];
  snprintf(tmp, sizeof(tmp), "GPU #%d %s (%dx%d cores) Grid(%dx%d)",
           gpuId, deviceProp.name, deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           nbThread / nbThreadPerGroup,
           nbThreadPerGroup);
  deviceName = std::string(tmp);

  // Prefer L1 cache for better performance (We do not use __shared__ memory at all)
  // Note: hipDeviceSetCacheConfig is deprecated since CUDA 9.0
  // Using modern per-function cache configuration instead
  err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(comp_kangaroos), hipFuncCachePreferL1);
  if (!CheckCudaError(err, "hipFuncSetCacheConfig for comp_kangaroos"))
  {
    // If per-function config fails, try the legacy device-wide setting for compatibility
    err = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    if (!CheckCudaError(err, "hipDeviceSetCacheConfig (fallback)"))
    {
      printf("Warning: Failed to set cache configuration, continuing with default settings\n");
      // Don't return here - cache config failure shouldn't prevent initialization
    }
  }

  // Allocate memory
  inputKangaroo = NULL;
  inputKangarooPinned = NULL;
  outputItem = NULL;
  outputItemPinned = NULL;
  jumpPinned = NULL;

  // Memory allocation with error handling
  // Input kangaroos
  kangarooSize = nbThread * GPU_GRP_SIZE * KSIZE * 8;
  err = hipMalloc((void **)&inputKangaroo, kangarooSize);
  if (!CheckCudaError(err, "hipMalloc(inputKangaroo)"))
  {
    CleanupOnConstructorFailure();
    return;
  }

  kangarooSizePinned = nbThreadPerGroup * GPU_GRP_SIZE * KSIZE * 8;
  err = hipHostAlloc(&inputKangarooPinned, kangarooSizePinned,
                      hipHostMallocWriteCombined | hipHostMallocMapped);
  if (!CheckCudaError(err, "hipHostAlloc(inputKangarooPinned)"))
  {
    CleanupOnConstructorFailure();
    return;
  }

  // OutputHash
  err = hipMalloc((void **)&outputItem, outputSize);
  if (!CheckCudaError(err, "hipMalloc(outputItem)"))
  {
    CleanupOnConstructorFailure();
    return;
  }

  err = hipHostAlloc(&outputItemPinned, outputSize, hipHostMallocMapped);
  if (!CheckCudaError(err, "hipHostAlloc(outputItemPinned)"))
  {
    CleanupOnConstructorFailure();
    return;
  }

  // Jump array
  jumpSize = NB_JUMP * 8 * 4;
  err = hipHostAlloc(&jumpPinned, jumpSize, hipHostMallocMapped);
  if (!CheckCudaError(err, "hipHostAlloc(jumpPinned)"))
  {
    CleanupOnConstructorFailure();
    return;
  }

  lostWarning = false;
  initialised = true;
  wildOffset.SetInt32(0);

#ifdef GPU_CHECK

  double minT = 1e9;
  for (int i = 0; i < 5; i++)
  {
    double t0 = Timer::get_tick();
    CUDA_LAUNCH_KERNEL(check_gpu, 1, 1, 0, 0);
    CudaErrorChecker::deviceSynchronize();
    double t1 = Timer::get_tick();
    if ((t1 - t0) < minT)
      minT = (t1 - t0);
  }
  printf("Cuda: %.3f ms\n", minT * 1000.0);
  exit(0);

#endif
}

GPUEngine::~GPUEngine()
{

  // Safe memory deallocation with modern error handling
  auto safeFree = [](auto &ptr, auto freeFunc, const char *name)
  {
    if (ptr)
    {
      hipError_t err = freeFunc(ptr);
      if (err != hipSuccess)
      {
        // Don't throw in destructor - just log warning
        printf("Warning - Failed to free %s: %s\n", name, hipGetErrorString(err));
      }
      ptr = nullptr;
    }
  };

  // Free GPU device memory
  safeFree(inputKangaroo, hipFree, "inputKangaroo");
  safeFree(outputItem, hipFree, "outputItem");

  // Free host pinned memory
  safeFree(inputKangarooPinned, hipHostFree, "inputKangarooPinned");
  safeFree(outputItemPinned, hipHostFree, "outputItemPinned");
  safeFree(jumpPinned, hipHostFree, "jumpPinned");

  // Force GPU context cleanup to prevent memory leaks
  // This ensures all GPU memory is properly released
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess)
  {
    printf("Warning - GPU device reset failed: %s\n", hipGetErrorString(err));
  }
}

int GPUEngine::GetMemory()
{
  return static_cast<int>(kangarooSize + outputSize + jumpSize);
}

int GPUEngine::GetGroupSize()
{
  return GPU_GRP_SIZE;
}

bool GPUEngine::GetGridSize(int gpuId, int *x, int *y)
{

  if (*x <= 0 || *y <= 0)
  {

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
      printf("GPUEngine: CudaGetDeviceCount %s\n", hipGetErrorString(error_id));
      return false;
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
      printf("GPUEngine: There are no available device(s) that support CUDA\n");
      return false;
    }

    if (gpuId >= deviceCount)
    {
      printf("GPUEngine::GetGridSize() Invalid gpuId\n");
      return false;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, gpuId);

    if (*x <= 0)
      *x = 2 * deviceProp.multiProcessorCount;
    if (*y <= 0)
      *y = 2 * _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    if (*y <= 0)
      *y = 128;
  }

  return true;
}

void *GPUEngine::AllocatePinnedMemory(size_t size)
{

  void *buff;

  hipError_t err = hipHostAlloc(&buff, size, hipHostMallocPortable);
  if (!CheckCudaError(err, "AllocatePinnedMemory"))
  {
    return NULL;
  }

  return buff;
}

void GPUEngine::FreePinnedMemory(void *buff)
{
  if (buff)
  {
    hipError_t err = hipHostFree(buff);
    CheckCudaError(err, "Warning - Failed to free pinned memory");
  }
}

void GPUEngine::ForceGPUCleanup()
{
  // Emergency GPU memory cleanup function
  // This can be called from signal handlers or exception handlers
  hipError_t err = hipDeviceReset();
  if (!CheckCudaError(err, "ForceGPUCleanup: GPU reset failed"))
  {
    // Error already logged
  }
  else
  {
    printf("ForceGPUCleanup: GPU memory successfully released\n");
  }
}

void GPUEngine::CleanupOnConstructorFailure()
{
  // Cleanup function to prevent memory leaks when constructor fails
  // Only cleanup resources that have been successfully allocated

  if (inputKangaroo)
  {
    hipFree(inputKangaroo);
    inputKangaroo = NULL;
  }

  if (inputKangarooPinned)
  {
    hipHostFree(inputKangarooPinned);
    inputKangarooPinned = NULL;
  }

  if (outputItem)
  {
    hipFree(outputItem);
    outputItem = NULL;
  }

  if (outputItemPinned)
  {
    hipHostFree(outputItemPinned);
    outputItemPinned = NULL;
  }

  if (jumpPinned)
  {
    hipHostFree(jumpPinned);
    jumpPinned = NULL;
  }

  initialised = false;
}

bool GPUEngine::CheckCudaStreamStatus()
{
  // Check the status of the default CUDA stream
  hipError_t err = hipStreamQuery(0); // 0 is the default stream

  if (err == hipSuccess)
  {
    return true; // Stream is idle (all operations completed)
  }
  else if (err == hipErrorNotReady)
  {
    return true; // Stream is busy but no error
  }
  else
  {
    // Stream has an error
    CheckCudaError(err, "CheckCudaStreamStatus: stream error detected");
    return false;
  }
}

bool GPUEngine::CheckGPUMemoryStatus()
{
  // Check GPU memory status and detect potential corruption
  size_t free_mem, total_mem;
  hipError_t err = hipMemGetInfo(&free_mem, &total_mem);

  if (!CheckCudaError(err, "CheckGPUMemoryStatus: hipMemGetInfo"))
  {
    return false;
  }

  // Check if we have sufficient memory (at least 10% free)
  if (free_mem < (total_mem / 10))
  {
    printf("Warning: GPU memory is critically low (Free: %zu MB, Total: %zu MB)\n",
           free_mem / (1024 * 1024), total_mem / (1024 * 1024));
  }

  // Verify our allocated memory is still valid by checking pointers
  if (initialised)
  {
    // Check if our device pointers are still valid
    hipPointerAttribute_t attr;
    err = hipPointerGetAttributes(&attr, inputKangaroo);
    if (err != hipSuccess)
    {
      CheckCudaError(err, "CheckGPUMemoryStatus: inputKangaroo pointer invalid");
      return false;
    }

    err = hipPointerGetAttributes(&attr, outputItem);
    if (err != hipSuccess)
    {
      CheckCudaError(err, "CheckGPUMemoryStatus: outputItem pointer invalid");
      return false;
    }
  }

  return true;
}

bool GPUEngine::CheckCudaContextStatus()
{
  // Check CUDA context status and device accessibility
  int currentDevice;
  hipError_t err = hipGetDevice(&currentDevice);
  if (!CheckCudaError(err, "CheckCudaContextStatus: hipGetDevice"))
  {
    return false;
  }

  // Verify device properties are still accessible
  hipDeviceProp_t prop;
  err = hipGetDeviceProperties(&prop, currentDevice);
  if (!CheckCudaError(err, "CheckCudaContextStatus: hipGetDeviceProperties"))
  {
    return false;
  }

  // Check if context is still valid by performing a simple operation
  void *testPtr;
  err = hipMalloc(&testPtr, 1);
  if (err == hipSuccess)
  {
    hipFree(testPtr); // Clean up test allocation
  }
  else
  {
    CheckCudaError(err, "CheckCudaContextStatus: context validation failed");
    return false;
  }

  return true;
}

// CudaMemoryGuard destructor implementation
CudaMemoryGuard::~CudaMemoryGuard()
{
  // Force cleanup of all GPU resources on destruction
  hipError_t err = hipDeviceReset();
  CheckCudaError(err, "CudaMemoryGuard: GPU cleanup warning");
}

void GPUEngine::PrintCudaInfo()
{

  hipError_t err;

  const char *sComputeMode[] =
      {
          "Multiple host threads",
          "Only one host thread",
          "No host thread",
          "Multiple process threads",
          "Unknown",
          NULL};

  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess)
  {
    CudaErrorHandler::HandleGPUEngineError(error_id, "CudaGetDeviceCount");
    return;
  }

  // This function call returns 0 if there are no CUDA capable devices.
  if (deviceCount == 0)
  {
    printf("GPUEngine: There are no available device(s) that support CUDA\n");
    return;
  }

  for (int i = 0; i < deviceCount; i++)
  {

    err = hipSetDevice(i);
    if (!CheckCudaError(err, ("hipSetDevice(" + std::to_string(i) + ")").c_str()))
    {
      return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, i);
    printf("GPU #%d %s (%dx%d cores) (Cap %d.%d) (%.1f MB) (%s)\n",
           i, deviceProp.name, deviceProp.multiProcessorCount,
           _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
           deviceProp.major, deviceProp.minor, (double)deviceProp.totalGlobalMem / 1048576.0,
           sComputeMode[deviceProp.computeMode]);
  }
}

int GPUEngine::GetNbThread()
{
  return nbThread;
}

// Helper function to calculate safe memory offset with bounds checking
inline size_t GPUEngine::CalculateKangarooOffset(int group, int thread, int coord_word) const
{
  // Validate inputs to prevent buffer overflows
  assert(group >= 0 && group < GPU_GRP_SIZE);
  assert(thread >= 0 && thread < nbThreadPerGroup);
  assert(coord_word >= 0 && coord_word < KSIZE);

  const int strideSize = nbThreadPerGroup * KSIZE;
  size_t offset = group * strideSize + thread + coord_word * nbThreadPerGroup;

  // Additional bounds check for the calculated offset
#ifdef DEBUG
  size_t maxOffset = kangarooSizePinned / sizeof(uint64_t);
  assert(offset < maxOffset);
#else
  // In release builds, we trust the input validation above
  // This avoids the unused variable warning while maintaining safety
  assert(offset < kangarooSizePinned / sizeof(uint64_t));
#endif

  return offset;
}

// Helper function to safely set coordinate data
inline void GPUEngine::SetCoordinateData(int group, int thread, int coord_base, const uint64_t *data, int word_count)
{
  for (int i = 0; i < word_count; i++)
  {
    size_t offset = CalculateKangarooOffset(group, thread, coord_base + i);
    inputKangarooPinned[offset] = data[i];
  }
}

// Helper function to safely get coordinate data
inline void GPUEngine::GetCoordinateData(int group, int thread, int coord_base, uint64_t *data, int word_count) const
{
  for (int i = 0; i < word_count; i++)
  {
    size_t offset = CalculateKangarooOffset(group, thread, coord_base + i);
    data[i] = inputKangarooPinned[offset];
  }
}

// Helper function to safely set single kangaroo coordinate to GPU memory
inline bool GPUEngine::SetSingleCoordinateToGPU(uint64_t kIdx, int coord_word, uint64_t value)
{
  // Calculate block, group, thread from kangaroo index
  int gSize = KSIZE * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * KSIZE;
  int blockSize = nbThreadPerGroup * gSize;

  uint64_t t = kIdx % nbThreadPerGroup;
  uint64_t g = (kIdx / nbThreadPerGroup) % GPU_GRP_SIZE;
  uint64_t b = kIdx / (nbThreadPerGroup * GPU_GRP_SIZE);

  // Validate indices
  if (b >= (uint64_t)(nbThread / nbThreadPerGroup) || g >= GPU_GRP_SIZE || t >= (uint64_t)nbThreadPerGroup)
  {
    printf("GPUEngine: SetSingleCoordinateToGPU: Invalid kangaroo index %llu\n", kIdx);
    return false;
  }

  // Calculate safe offset
  size_t offset = b * blockSize + g * strideSize + t + coord_word * nbThreadPerGroup;

  // Copy single value to GPU
  inputKangarooPinned[0] = value;
  hipError_t err = hipMemcpy(inputKangaroo + offset, inputKangarooPinned, sizeof(uint64_t), hipMemcpyHostToDevice);

  return CheckCudaError(err, "SetSingleCoordinateToGPU", false, "kangaroo", static_cast<int>(kIdx));
}

// Optimized batch transfer function for single kangaroo - eliminates multiple small hipMemcpy calls
inline bool GPUEngine::SetKangarooBatch(uint64_t kIdx, Int *px, Int *py, Int *d)
{
  // Calculate block, group, thread from kangaroo index
  int gSize = KSIZE * GPU_GRP_SIZE;
  int strideSize = nbThreadPerGroup * KSIZE;
  int blockSize = nbThreadPerGroup * gSize;

  uint64_t t = kIdx % nbThreadPerGroup;
  uint64_t g = (kIdx / nbThreadPerGroup) % GPU_GRP_SIZE;
  uint64_t b = kIdx / (nbThreadPerGroup * GPU_GRP_SIZE);

  // Validate indices
  if (b >= (uint64_t)(nbThread / nbThreadPerGroup) || g >= GPU_GRP_SIZE || t >= (uint64_t)nbThreadPerGroup)
  {
    printf("GPUEngine: SetKangarooBatch: Invalid kangaroo index %llu\n", kIdx);
    return false;
  }

  // Prepare distance with wild offset
  Int dOff;
  dOff.Set(d);
  if (kIdx % 2 == WILD)
    dOff.ModAddK1order(&wildOffset);

  // Batch all coordinate updates into a single buffer
  // Layout: [X0,X1,X2,X3,Y0,Y1,Y2,Y3,D0,D1,JUMP]
  uint64_t batchData[11];

  // X coordinate (4 words)
  for (int i = 0; i < 4; i++)
    batchData[i] = px->bits64[i];

  // Y coordinate (4 words)
  for (int i = 0; i < 4; i++)
    batchData[4 + i] = py->bits64[i];

  // Distance (2 words)
  batchData[8] = dOff.bits64[0];
  batchData[9] = dOff.bits64[1];

#ifdef USE_SYMMETRY
  // Last jump count (1 word)
  batchData[10] = (uint64_t)NB_JUMP;
  int totalWords = 11;
#else
  int totalWords = 10;
#endif

  // Use strided memory copy for optimal GPU memory layout
  size_t baseOffset = b * blockSize + g * strideSize + t;

  // Prepare batch data for strided memory transfer
  for (int i = 0; i < totalWords; i++)
  {
    inputKangarooPinned[i] = batchData[i];
  }

  // Single batched memory transfer instead of multiple small transfers
  hipError_t err = hipMemcpy2D(
      inputKangaroo + baseOffset,          // dst
      nbThreadPerGroup * sizeof(uint64_t), // dpitch (stride between elements)
      inputKangarooPinned,                 // src
      sizeof(uint64_t),                    // spitch (contiguous source)
      sizeof(uint64_t),                    // width (element size)
      totalWords,                          // height (number of elements)
      hipMemcpyHostToDevice               // kind
  );

  return CheckCudaError(err, "SetKangarooBatch", false, "kangaroo", static_cast<int>(kIdx));
}

void GPUEngine::SetKangaroos(Int *px, Int *py, Int *d)
{

  // Sets the kangaroos of each thread
  // Using safer approach with helper functions to eliminate manual index calculations
  int gSize = KSIZE * GPU_GRP_SIZE;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;
  int idx = 0;

  for (int b = 0; b < nbBlock; b++)
  {
    for (int g = 0; g < GPU_GRP_SIZE; g++)
    {
      for (int t = 0; t < nbThreadPerGroup; t++)
      {

        // X coordinate (4 words: 0-3)
        SetCoordinateData(g, t, 0, px[idx].bits64, 4);

        // Y coordinate (4 words: 4-7)
        SetCoordinateData(g, t, 4, py[idx].bits64, 4);

        // Distance (2 words: 8-9)
        Int dOff;
        dOff.Set(&d[idx]);
        if (idx % 2 == WILD)
          dOff.ModAddK1order(&wildOffset);
        SetCoordinateData(g, t, 8, dOff.bits64, 2);

#ifdef USE_SYMMETRY
        // Last jump (1 word: 10)
        uint64_t jumpData = (uint64_t)NB_JUMP;
        SetCoordinateData(g, t, 10, &jumpData, 1);
#endif

        idx++;
      }
    }

    uint32_t offset = b * blockSize;
    hipError_t err = hipMemcpy(inputKangaroo + offset, inputKangarooPinned, kangarooSizePinned, hipMemcpyHostToDevice);
    if (!CheckCudaError(err, "SetKangaroos: hipMemcpy block transfer", false, "block", b))
    {
      return; // Early return on memory transfer failure
    }
  }

  // Final error check for any remaining CUDA errors
  hipError_t finalErr = hipGetLastError();
  CheckCudaError(finalErr, "SetKangaroos: final check");
}

void GPUEngine::GetKangaroos(Int *px, Int *py, Int *d)
{

  if (inputKangarooPinned == NULL)
  {
    printf("GPUEngine: GetKangaroos: Cannot retreive kangaroos, mem has been freed\n");
    return;
  }

  // Gets the kangaroos of each thread
  // Using safer approach with helper functions to eliminate manual index calculations
  int gSize = KSIZE * GPU_GRP_SIZE;
  int nbBlock = nbThread / nbThreadPerGroup;
  int blockSize = nbThreadPerGroup * gSize;
  int idx = 0;

  for (int b = 0; b < nbBlock; b++)
  {

    uint32_t offset = b * blockSize;
    hipMemcpy(inputKangarooPinned, inputKangaroo + offset, kangarooSizePinned, hipMemcpyDeviceToHost);

    for (int g = 0; g < GPU_GRP_SIZE; g++)
    {

      for (int t = 0; t < nbThreadPerGroup; t++)
      {

        // X coordinate (4 words: 0-3)
        GetCoordinateData(g, t, 0, px[idx].bits64, 4);
        px[idx].bits64[4] = 0; // Clear the 5th word

        // Y coordinate (4 words: 4-7)
        GetCoordinateData(g, t, 4, py[idx].bits64, 4);
        py[idx].bits64[4] = 0; // Clear the 5th word

        // Distance (2 words: 8-9)
        Int dOff;
        dOff.SetInt32(0);
        GetCoordinateData(g, t, 8, dOff.bits64, 2);
        if (idx % 2 == WILD)
          dOff.ModSubK1order(&wildOffset);
        d[idx].Set(&dOff);

        idx++;
      }
    }
  }

  hipError_t err = hipGetLastError();
  CheckCudaError(err, "GetKangaroos");
}

void GPUEngine::SetKangaroo(uint64_t kIdx, Int *px, Int *py, Int *d)
{
  // PERFORMANCE OPTIMIZATION: Use batch transfer instead of multiple small hipMemcpy calls
  // This eliminates the inefficient pattern of 11 separate GPU memory transfers per kangaroo

  // Use optimized batch transfer function
  SetKangarooBatch(kIdx, px, py, d);

  // Legacy fallback method (kept for debugging/compatibility if needed)
  // The old method made 11 separate hipMemcpy calls which is highly inefficient:
  // - 4 calls for X coordinate
  // - 4 calls for Y coordinate
  // - 2 calls for distance
  // - 1 call for jump count (if USE_SYMMETRY)
  //
  // The new batch method reduces this to a single strided hipMemcpy2D call
}

bool GPUEngine::callKernel()
{

  // Reset nbFound with error checking
  hipError_t err = hipMemset(outputItem, 0, 4);
  if (!CheckCudaError(err, "callKernel: hipMemset"))
  {
    return false;
  }

  // Call the kernel (Perform STEP_SIZE keys per thread)
  comp_kangaroos<<<nbThread / nbThreadPerGroup, nbThreadPerGroup>>>(inputKangaroo, maxFound, outputItem, dpMask);

  // Check for kernel launch errors
  err = hipGetLastError();
  if (!CheckCudaError(err, "callKernel: kernel launch", true))
  {
    return false;
  }

// For critical operations, ensure kernel execution completes successfully
// Note: This is optional for performance reasons, but provides better error detection
#ifdef CUDA_STRICT_ERROR_CHECKING
  err = hipDeviceSynchronize();
  if (!CheckCudaError(err, "callKernel: device synchronization", true))
  {
    return false;
  }
#endif

  return true;
}

void GPUEngine::SetParams(uint64_t dpMask, Int *distance, Int *px, Int *py)
{

  this->dpMask = dpMask;

  for (int i = 0; i < NB_JUMP; i++)
    memcpy(jumpPinned + 2 * i, distance[i].bits64, 16);
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(jD), jumpPinned, jumpSize / 2);
  if (!CheckCudaError(err, "SetParams: Failed to copy jD to constant memory"))
  {
    return;
  }

  for (int i = 0; i < NB_JUMP; i++)
    memcpy(jumpPinned + 4 * i, px[i].bits64, 32);
  err = hipMemcpyToSymbol(HIP_SYMBOL(jPx), jumpPinned, jumpSize);
  if (!CheckCudaError(err, "SetParams: Failed to copy jPx to constant memory"))
  {
    return;
  }

  for (int i = 0; i < NB_JUMP; i++)
    memcpy(jumpPinned + 4 * i, py[i].bits64, 32);
  err = hipMemcpyToSymbol(HIP_SYMBOL(jPy), jumpPinned, jumpSize);
  if (!CheckCudaError(err, "SetParams: Failed to copy jPy to constant memory"))
  {
    return;
  }
}

bool GPUEngine::callKernelAndWait()
{

  // Debug function with proper error handling
  if (!callKernel())
  {
    return false; // callKernel already logged the error
  }

  hipError_t err = hipMemcpy(outputItemPinned, outputItem, outputSize, hipMemcpyDeviceToHost);
  if (!CheckCudaError(err, "callKernelAndWait: hipMemcpy", true))
  {
    return false;
  }

  // Final error check
  err = hipGetLastError();
  return CheckCudaError(err, "callKernelAndWait: final check", true);
}

bool GPUEngine::Launch(std::vector<ITEM> &hashFound, bool spinWait)
{

  hashFound.clear();

  // Perform comprehensive CUDA error checking before launch
  if (!CheckCudaStreamStatus())
  {
    printf("Launch: CUDA stream error detected before operation\n");
    return false;
  }

  if (!CheckGPUMemoryStatus())
  {
    printf("Launch: GPU memory error detected before operation\n");
    return false;
  }

  // Get the result

  if (spinWait)
  {
    // Synchronous memory copy with error checking
    hipError_t err = hipMemcpy(outputItemPinned, outputItem, outputSize, hipMemcpyDeviceToHost);
    if (!CheckCudaError(err, "Launch: synchronous hipMemcpy"))
    {
      return false;
    }
  }
  else
  {
    // Use hipMemcpyAsync to avoid default spin wait of hipMemcpy which takes 100% CPU
    hipEvent_t evt;
    hipError_t err = hipEventCreate(&evt);
    if (!CheckCudaError(err, "Launch: hipEventCreate"))
    {
      return false;
    }

    err = hipMemcpyAsync(outputItemPinned, outputItem, 4, hipMemcpyDeviceToHost, 0);
    if (!CheckCudaError(err, "Launch: hipMemcpyAsync"))
    {
      hipEventDestroy(evt); // Cleanup on error
      return false;
    }

    err = hipEventRecord(evt, 0);
    if (!CheckCudaError(err, "Launch: hipEventRecord"))
    {
      hipEventDestroy(evt); // Cleanup on error
      return false;
    }

    // Wait for async operation to complete with error checking
    hipError_t queryResult;
    while ((queryResult = hipEventQuery(evt)) == hipErrorNotReady)
    {
      // Sleep 1 ms to free the CPU
      Timer::SleepMillis(1);
    }

    // Check if the event query failed for reasons other than "not ready"
    if (queryResult != hipSuccess)
    {
      CheckCudaError(queryResult, "Launch: hipEventQuery");
      hipEventDestroy(evt); // Cleanup on error
      return false;
    }

    err = hipEventDestroy(evt);
    if (!CheckCudaError(err, "Launch: hipEventDestroy"))
    {
      return false;
    }
  }

  hipError_t err = hipGetLastError();
  if (!CheckCudaError(err, "Launch"))
  {
    return false;
  }

  // Look for prefix found
  uint32_t nbFound = outputItemPinned[0];
  if (nbFound > maxFound)
  {
    // prefix has been lost
    uint32_t itemsLost = nbFound - maxFound;
    if (!lostWarning)
    {
      printf("\n[!] WARNING: %u distinguished points lost!\n", itemsLost);
      printf("[*] Solutions:\n");
      printf("   1. Reduce GPU threads: -g <smaller_x>,<smaller_y>\n");
      printf("   2. Increase DP size: -d <larger_value>\n");
      printf("   3. Current buffer size: %u items\n", maxFound);
      printf("   4. Detected overflow: %u items\n", nbFound);
      lostWarning = true;
    }

    // Log periodic warnings for severe overflow
    static uint32_t warningCounter = 0;
    warningCounter++;
    if (warningCounter % 100 == 0)
    {
      printf("[!] Overflow continues: %u items lost (warning #%u)\n", itemsLost, warningCounter);
    }

    nbFound = maxFound;
  }

  // When can perform a standard copy, the kernel is eneded
  hipMemcpy(outputItemPinned, outputItem, nbFound * ITEM_SIZE + 4, hipMemcpyDeviceToHost);

  for (uint32_t i = 0; i < nbFound; i++)
  {
    uint32_t *itemPtr = outputItemPinned + (i * ITEM_SIZE32 + 1);
    ITEM it;

    it.kIdx = *((uint64_t *)(itemPtr + 12));

    uint64_t *x = (uint64_t *)itemPtr;
    it.x.bits64[0] = x[0];
    it.x.bits64[1] = x[1];
    it.x.bits64[2] = x[2];
    it.x.bits64[3] = x[3];
    it.x.bits64[4] = 0;

    uint64_t *d = (uint64_t *)(itemPtr + 8);
    it.d.bits64[0] = d[0];
    it.d.bits64[1] = d[1];
    it.d.bits64[2] = 0;
    it.d.bits64[3] = 0;
    it.d.bits64[4] = 0;
    if (it.kIdx % 2 == WILD)
      it.d.ModSubK1order(&wildOffset);

    hashFound.push_back(it);
  }

  return callKernel();
}
